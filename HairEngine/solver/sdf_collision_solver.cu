
#include <hip/hip_runtime.h>

#ifdef HAIRENGINE_ENABLE_CUDA

#include <cstdio>
#include <cuda_runtime.h>
#include <device_functions.h>
#include "../util/helper_math.cuh"


__global__
void SDFCollisionSolver_cudaComputeVelocitiesKernal(const float3 *prePoses,
                                                    const float3 *poses, const int3 *indices, float3 *outVel, float tInv, int nprim) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nprim)
		return;

	int3 pids = indices[i];

	float3 p0 = (1.0f / 3.0f) * (prePoses[pids.x] + prePoses[pids.y] + prePoses[pids.z] );
	float3 p1 = (1.0f / 3.0f) * (poses[pids.x] + poses[pids.y] + poses[pids.z] );

	outVel[i] = (p1 - p0) * tInv;
}

void SDFCollisionSolver_cudaComputeVelocities(const float3 *prePoses,
                                              const float3 *poses, const int3 *indices, float3 *outVel,
                                              float tInv, int nprim, int nblock, int nthread) {
	SDFCollisionSolver_cudaComputeVelocitiesKernal<<<nblock, nthread>>>(prePoses, poses, indices, outVel, tInv, nprim);
	cudaDeviceSynchronize();

}

__global__
void SDFCollisionSolver_cudaComputeSDFGridKernal(const float3 *poses,
                                                 const int3 *indices, unsigned long long * outGrid,
                                                 int npoint, int nprim, int3 n, int margin, float3 origin, float3 d) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nprim)
		return;

	int nyz = n.y * n.z;
	int3 margin3 { margin, margin, margin };
	float3 dInv { 1.0f / d.x, 1.0f / d.y, 1.0f / d.z };

	float3 p[3] {poses[indices[i].x], poses[indices[i].y], poses[indices[i].z]};

	// Compute the bounding box
	float3 bboxMin = p[0];
	float3 bboxMax = p[0];

	bboxMin = fminf(bboxMin, p[1]);
	bboxMin = fminf(bboxMin, p[2]);

	bboxMax = fmaxf(bboxMax, p[1]);
	bboxMax = fmaxf(bboxMax, p[2]);

	int3 minIndex = max(make_int3((bboxMin - origin) * dInv) - margin3, make_int3(0));
	int3 maxIndex = min(make_int3((bboxMax - origin) * dInv) + margin3 + make_int3(1), n);

	for (int ix = minIndex.x; ix <= maxIndex.x; ++ix)
		for (int iy = minIndex.y; iy <= maxIndex.y; ++iy)
			for (int iz = minIndex.z; iz <= maxIndex.z; ++iz) {

				float3 pos = origin + make_float3(ix, iy, iz) * d;

				float signedDist = cudaPointToTriangleSignedDistance(pos, p[0], p[1], p[2]);

				unsigned long long pack;

				// Pack the signed distance to the most significant 32 bit
				// and the primitive index (i) to the last significant 32 bit
				// Currently CUDA is not supported bit shift in 64-bit, so the direct way is to assign the the data
				// directly into the address. Make sure that it is Little-Endian
				uint32_t *packAddr = reinterpret_cast<uint32_t*>(&pack);
				packAddr[1] = floatflip(signedDist);
				packAddr[0] = static_cast<uint32_t>(i);


				int offset = ix * nyz + iy * n.z + iz;
				atomicMin(outGrid + offset, pack);
			}
}

void SDFCollisionSolver_cudaComputeSDFGrid(const float3 *poses, const int3 *indices, unsigned long long * outGrid,
                                                 int npoint, int nprim, int3 n, int margin, float3 origin, float3 d, int nblock, int nthread) {
	SDFCollisionSolver_cudaComputeSDFGridKernal<<<nblock, nthread>>>(poses, indices, outGrid, npoint, nprim, n, margin, origin, d);
	cudaDeviceSynchronize();
}

__device__ inline bool SDFCollisionSolver_querySDF(float3 pos, const unsigned long long *grid, const float3 *vels, float3 origin, float3 d, float3 dInv, int3 n, float *outDist, float3 *outGradient, float3 *outV = nullptr) {

	if ((pos.x < origin.x) || (pos.y < origin.y) || (pos.z < origin.z))
		return false;

	int3 maxcoor = { n.x - 1, n.y - 1, n.z - 1 };
	float3 bboxMax = origin + d * make_float3(maxcoor);

	if ((pos.x > bboxMax.x) || (pos.y > bboxMax.y) || (pos.z > bboxMax.z))
		return false;

	float3 index3f = (pos - origin) * dInv;
	int3 index3 = min( make_int3(index3f), maxcoor);

	int nyz = n.y * n.z;

	int offsets[8];
	offsets[0] = index3.x * nyz + index3.y * (n.z) + index3.z;
	offsets[1] = offsets[0] + 1;
	offsets[2] = offsets[0] + n.z;
	offsets[3] = offsets[0] + n.z + 1;
	offsets[4] = offsets[0] + nyz;
	offsets[5] = offsets[4] + 1;
	offsets[6] = offsets[4] + n.z;
	offsets[7] = offsets[4] + n.z + 1;

	float nodesSignedDist[8];
	int nodesPrimIdx[8];
	for (int i = 0; i < 8; ++i) {
		unsigned long long pack = grid[offsets[i]];
		uint32_t *packAddr = reinterpret_cast<uint32_t *>(&pack);

		nodesSignedDist[i] = ifloatflip(packAddr[1]); // Most significant 32-bit
		nodesPrimIdx[i] = static_cast<int>(packAddr[0]); // Least significant 32-bit
	}

	bool isAnyCellInvalid = false;
	bool isAllCellLargerThanContour = true;

	for (int i = 0; i < 8; ++i) {
		isAnyCellInvalid |= (nodesPrimIdx[i] == 0xffffffff);
		isAllCellLargerThanContour &= nodesSignedDist[i] > 0.0f;
	}

	if (isAnyCellInvalid || isAllCellLargerThanContour)
		return false;

	float3 t = index3f - make_float3(index3);

	float signedDist = 0.0f;
	float3 gradient = { 0.0f, 0.0f, 0.0f };

	if (outV) {
		*outV = { 0.0f, 0.0f, 0.0f };
	}

	for (int i = 0; i < 8; ++i) {

		float cx = (i & 4) ? t.x : 1.0f - t.x;
		float cy = (i & 2) ? t.y : 1.0f - t.y;
		float cz = (i & 1) ? t.z : 1.0f - t.z;

		signedDist += nodesSignedDist[i] * (cx * cy * cz);

		if (outV) {
			(*outV) += vels[nodesPrimIdx[i]] * (cx * cy * cz);
		}

		gradient.x += ((i & 4) ? 1.0f : -1.0f) * cy * cz * nodesSignedDist[i];
		gradient.y += ((i & 2) ? 1.0f : -1.0f) * cx * cz * nodesSignedDist[i];
		gradient.z += ((i & 1) ? 1.0f : -1.0f) * cx * cy * nodesSignedDist[i];
	}

	*outDist = signedDist;
	*outGradient = gradient;

	return true;
}

__global__
void SDFCollisionSolver_cudaResolveCollisionKernal(float3 *parPoses, float3 *parVels, const unsigned char *parLocalIndices, const float3 * vels,
                                                   const unsigned long long *grid, int npar, float3 origin, float3 d, int3 n, float time, float fraction, bool changeHairRoot) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= npar)
		return;

	bool unchanged = false;

	// Check whether to change hair root
	if (!changeHairRoot && parLocalIndices[i] == 0)
		unchanged = true;

	float3 vp = parVels[i];
	float3 pos = parPoses[i]; // Estimated position

	float3 dInv { 1.0f / d.x, 1.0f / d.y, 1.0f / d.z };
//	int3 maxcoor = n - 1;
//
//	if ((pos.x < origin.x) || (pos.y < origin.y) || (pos.z < origin.z))
//		return;
//
//	float3 bboxMax = origin + d * make_float3(maxcoor);
//	if ((pos.x > bboxMax.x) || (pos.y > bboxMax.y) || (pos.z > bboxMax.z))
//		return;
//
//	float3 index3f = (pos - origin) * dInv;
//	int3 index3 = min( make_int3(index3f), maxcoor);
//
//	int nyz = n.y * n.z;
//
//	int offsets[8];
//	offsets[0] = index3.x * nyz + index3.y * (n.z) + index3.z;
//	offsets[1] = offsets[0] + 1;
//	offsets[2] = offsets[0] + n.z;
//	offsets[3] = offsets[0] + n.z + 1;
//	offsets[4] = offsets[0] + nyz;
//	offsets[5] = offsets[4] + 1;
//	offsets[6] = offsets[4] + n.z;
//	offsets[7] = offsets[4] + n.z + 1;
//
//	float nodesSignedDist[8];
//	int nodesPrimIdx[8];
//	for (int i = 0; i < 8; ++i) {
//		unsigned long long pack = grid[offsets[i]];
//		uint32_t *packAddr = reinterpret_cast<uint32_t *>(&pack);
//
//		nodesSignedDist[i] = ifloatflip(packAddr[1]); // Most significant 32-bit
//		nodesPrimIdx[i] = static_cast<int>(packAddr[0]); // Least significant 32-bit
//	}
//
//	bool isAnyCellInvalid = false;
//	bool isAllCellLargerThanContour = true;
//
//	for (int i = 0; i < 8; ++i) {
//		isAnyCellInvalid |= (nodesPrimIdx[i] == 0xffffffff);
//		isAllCellLargerThanContour &= nodesSignedDist[i] > contour;
//	}
//
//	if (isAnyCellInvalid || isAllCellLargerThanContour)
//		return;
//
//	float3 t = index3f - make_float3(index3);
//	float chunkx[2] { 1 - t.x, t.x };
//	float chunky[2] { 1 - t.y, t.y };
//	float chunkz[2] { 1 - t.z, t.z };
//
//	float signedDist = 0.0f;
//
//	for (int i = 0; i < 8; ++i) {
//		int boolx = i >> 2, booly = (i >> 1) & 1, boolz = i & 1;
//		signedDist += nodesSignedDist[i] * (chunkx[boolx] * chunky[booly] * chunkz[boolz]);
//	}
//
//	if (signedDist > contour)
//		return;
//
//	float3 gradient = make_float3(0.0f);
//	float3 v = make_float3(0.0f); // Object velocity
//
//	for (int i = 0; i < 8; ++i) {
//		int boolx = i >> 2, booly = (i >> 1) & 1, boolz = i & 1;
//		float cx = chunkx[boolx], cy = chunky[booly], cz = chunkz[boolz];
//
//		v += vels[nodesPrimIdx[i]] * (cx * cy * cz);
//
//		gradient += make_float3(boolx ? 1.0f : -1.0f, booly ? 1.0f : -1.0f, boolz ? 1.0f : -1.0f)
//		            * make_float3(cy * cz, cx * cz, cx * cy)
//		            * nodesSignedDist[i];
//	}
//
//	// Normalize
//	gradient *= dInv;
//	gradient /= length(gradient);
//
//	float3 vpt, vpn, vt, vn;
//	vpn = gradient * dot(vp, gradient);
//	vpt = vp - vpn;
//	vn = gradient * dot(v, gradient);
//	vt = v - vn;
//
//	float3 vrelt = vpt - vt;
//	vrelt *= fmaxf(0.0f, 1.0f - fraction * length(vpn - vn) / length(vrelt));
//	//vrelt = { 0.0f, 0.0f, 0.0f };
//
//	vpt = vt + vrelt;
//	vpn = vn;
//
//	vp = vpt + vpn;

	// Write back
	//parVels[i] = { 0.0f, 0.0f, 0.0f };
	//parPoses[i] = pos + (contour - signedDist) * gradient;

	float signedDist;
	float3 gradient;
	float3 v;

	if (!SDFCollisionSolver_querySDF(pos + time * vp, grid, vels, origin, d, dInv, n, &signedDist, &gradient, &v))
		unchanged = true;

	if (signedDist > 0.0f)
		unchanged = true;

	if (unchanged) {
		parPoses[i] = pos + time * vp;
		parVels[i] = vp;
		return;
	}

	// Fix the velocity
	gradient /= length(gradient);
	float3 vpt, vpn, vt, vn;
	vpn = gradient * dot(vp, gradient);
	vpt = vp - vpn;
	vn = gradient * dot(v, gradient);
	vt = v - vn;

	float3 vrelt = vpt - vt;
	vrelt *= fmaxf(0.0f, 1.0f - fraction * length(vpn - vn) / length(vrelt));

	vp = vt + vrelt + vn;
//	vp = {0.0f, 0.0f, 0.0f};
	pos += vp * time;

	if (SDFCollisionSolver_querySDF(pos, grid, vels, origin, d, dInv, n, &signedDist, &gradient)) {
		pos -= signedDist * gradient / length(gradient);
	}

	parPoses[i] = pos;
	parVels[i] = vp;
}

void SDFCollisionSolver_cudaResolveCollision(float3 *parPoses, float3 *parVels, const unsigned char *parLocalIndices, const float3 * vels,
                                                   const unsigned long long *grid, int npar, float3 origin, float3 d, int3 n, float time, float fraction, bool changeHairRoot, int nblock, int nthread) {
	SDFCollisionSolver_cudaResolveCollisionKernal<<<nblock, nthread>>>(parPoses, parVels, parLocalIndices, vels, grid, npar, origin, d, n, time, fraction, changeHairRoot);
	cudaDeviceSynchronize();
}

#endif