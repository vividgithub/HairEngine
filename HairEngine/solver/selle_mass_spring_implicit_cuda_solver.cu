#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "../util/simple_mat.h"

namespace HairEngine {

	__device__ __forceinline__
	void getSpringInfo(float3 p1, float3 p2, float3 rp1, float3 rp2, float t, float k, float3 &impulse, Mat3 &dm) {
		float3 d = p2 - p1;
		float l = length(d);
		d /= l;

		float f = t * t * k;

		impulse = (k * (l - length(rp1 - rp2)) * t) * d;

		dm.at<0, 0>() = f * d.x * d.x;
		dm.at<0, 1>() = dm.at<1, 0>() = f * d.x * d.y;
		dm.at<0, 2>() = dm.at<2, 0>() = f * d.x * d.z;
		dm.at<1, 1>() = f * d.y * d.y;
		dm.at<1, 2>() = dm.at<2, 1>() = f * d.y * d.z;
		dm.at<2, 2>() = f * d.z * d.z;
	}

	__global__
	void SelleMassSpringImplicitCudaSolver_resolveStrandDynamicsKernal(
			Mat3 **A,
			Mat3 **L,
			Mat3 **U,
			float3 *y,
			float3 *b,
			float3 *poses,
			float3 *prevPoses,
			float3 *restPoses,
			float3 *vels,
			float3 *impulses,
			const float *rigidness,
			Mat3 dTransform,
			float3 dTranslation,
			int numParticle,
			int numStrand,
			int numParticlePerStrand,
			float pmass,
			float damping,
			float kStretch,
			float kBending,
			float kTorsion,
			float strainLimitingTolerance,
			float t
	) {
		// One for each strand
		int si = blockIdx.x * blockDim.x + threadIdx.x;
		if (si >= numStrand)
			return;

		// Alias
		const auto & n = numParticlePerStrand;

		// Initialize the b and A
		float3 p[4]; // Previous poses, store in register
		float3 rp[4]; // Previous rest poses, store in register

		float3 impulse;
		Mat3 dm;
		for (int i = si, li = 0; i < numParticle; i += numStrand, ++li) {

			A[3][i] = Mat3::Diagnoal(pmass + damping * t);
			A[0][i] = A[1][i] = A[2][i] = A[4][i] = A[5][i] = A[6][i] = Mat3::Zero();

			b[i] = pmass * vels[i] + impulses[i] * t;

			// Apply spring force, since b[pi] and A[xxx][pi] has been correctly initialized, it is safe to modify
			p[3] = prevPoses[i];
			rp[3] = restPoses[i];

			// Stretch spring
			if (li >= 1) {
				getSpringInfo(p[2], p[3], rp[2], rp[3], t, kStretch, impulse, dm);

				auto pi = i - numStrand;
				b[pi] += impulse;
				b[i] -= impulse;

				A[3][pi] += dm;
				A[3][i] += dm;
				A[4][pi] -= dm;
				A[2][i] -= dm;
			}

//			if (si == 0) {
//				printf("p[2]: {%f, %f, %f}, p[3]: {%f, %f, %f}, rp[2]: {%f, %f, %f}, rp[3]: {%f, %f, %f}\n", p[2].x, p[2].y, p[2].z, p[3].x, p[3].y, p[3].z, rp[2].x, rp[2].y, rp[2].z, rp[3].x, rp[3].y, rp[3].z);
//				printf("impulse: {%f, %f, %f}\n", impulse.x, impulse.y, impulse.z);
//				printf("dm: "); dm.print(); printf("\n");
//			}
//
			// Bending spring
			if (li >= 2) {
				getSpringInfo(p[1], p[3], rp[1], rp[3], t, kBending, impulse, dm);

				auto pi = i - 2 * numStrand;
				b[pi] += impulse;
				b[i] -= impulse;

				A[3][pi] += dm;
				A[3][i] += dm;
				A[5][pi] -= dm;
				A[1][i] -= dm;
			}

			// Torsion spring
			if (li >= 3) {
				getSpringInfo(p[0], p[3], rp[0], rp[3], t, kTorsion, impulse, dm);

				auto pi = i - 3 * numStrand;
				b[pi] += impulse;
				b[i] -= impulse;

				A[3][pi] += dm;
				A[3][i] += dm;
				A[6][pi] -= dm;
				A[0][i] -= dm;
			}

			p[0] = p[1]; p[1] = p[2]; p[2] = p[3];
			rp[0] = rp[1]; rp[1] = rp[2]; rp[2] = rp[3];
		}

		// Initialize b and A of the strand root
		b[si] = (dTransform * prevPoses[si] + dTranslation - prevPoses[si]) / t;
		A[0][si] = A[1][si] = A[2][si] = A[4][si] = A[5][si] = A[6][si] = Mat3::Zero();
		A[3][si] = Mat3::Identity();

		// Heptadignoal solver
		int i1, i2, i3;

		for (int i = si, li = 0; i < numParticle; i += numStrand, ++li) {

			i1 = i - numStrand;
			i2 = i1 - numStrand;
			i3 = i2 - numStrand;

			//compute L3
			L[3][i] = li >= 3 ? A[0][i] : Mat3::Zero();

			//compute L2
			L[2][i] = li >= 2 ? A[1][i] : Mat3::Zero();
			if (li >= 3)
				L[2][i] -= L[3][i] * U[0][i3];

			//compute L1
			L[1][i] = li >= 1 ? A[2][i] : Mat3::Zero();
			if (li >= 2)
				L[1][i] -= L[2][i] * U[0][i2];
			if (li >= 3)
				L[1][i] -= L[3][i] * U[1][i3];

			//compute L0
			L[0][i] = A[3][i];
			if (li >= 1)
				L[0][i] -= L[1][i] * U[0][i1];
			if (li >= 2)
				L[0][i] -= L[2][i] * U[1][i2];
			if (li >= 3)
				L[0][i] -= L[3][i] * U[2][i3];

			//compute L0i
			L[4][i] = L[0][i].inverse();

			//compute U2
			U[2][i] = li + 3 < n ? L[4][i] * A[6][i] : Mat3::Zero();

			//compute U1
			U[1][i] = li + 2 < n ? A[5][i] : Mat3::Zero();
			if (li >= 1)
				U[1][i] -= L[1][i] * U[2][i1];
			U[1][i] = L[4][i] * U[1][i];

			//compute U0
			U[0][i] = li + 1 < n ? A[4][i] : Mat3::Zero();
			if (li >= 1)
				U[0][i] -= L[1][i] * U[1][i1];
			if (li >= 2)
				U[0][i] -= L[2][i] * U[2][i2];
			U[0][i] = L[4][i] * U[0][i];

			//compute y
			y[i] = b[i];
			if (li >= 1)
				y[i] -= L[1][i] * y[i1];
			if (li >= 2)
				y[i] -= L[2][i] * y[i2];
			if (li >= 3)
				y[i] -= L[3][i] * y[i3];
			y[i] = L[4][i] * y[i];
		}

		// Compute the final velocity and poses
		for (int i = numParticle - numStrand + si, li = n - 1; i >= 0; i -= numStrand, --li) {

			i1 = i + numStrand;
			i2 = i1 + numStrand;
			i3 = i2 + numStrand;

			vels[i] = y[i];
			if (li + 1 < n)
				vels[i] -= U[0][i] * vels[i1];
			if (li + 2 < n)
				vels[i] -= U[1][i] * vels[i2];
			if (li + 3 < n)
				vels[i] -= U[2][i] * vels[i3];

			float3 prevPos = prevPoses[i];

			// Rigidness
			poses[i] = lerp(prevPos + vels[i] * t, dTransform * prevPos + dTranslation, rigidness[i]);
		}

		// Apply strain limiting
		if (strainLimitingTolerance > 1.0f) {
			p[0] = poses[si];
			rp[0] = restPoses[si];
			for (int i = si + numStrand; i < numParticle; i += numStrand) {
				p[1] = poses[i];
				rp[1] = restPoses[i];

				float ltol = length(rp[1] - rp[0]) * strainLimitingTolerance;

				float3 d = p[1] - p[0];
				float l = length(d);

				if (l > ltol)
					p[1] = p[0] + d * (ltol / l);

				// Write back
				poses[i] = p[1];

				// Assign for next itereation
				p[0] = p[1];
				rp[0] = rp[1];
			}
		}

//		if (si == 0) {
//			printf("Initialize A and b:\n");
//			for (int i = si, li = 0; i < numParticle; i += numStrand, ++li) {
//
//				printf("%d(%d)\n", i, li);
//
//				for (int k = 0; k < 7; ++k) {
//					printf("\t\tA[%d][%d]: ", k, i);
//					A[k][i].print();
//					printf("\n");
//				}
//
//				for (int k = 0; k < 5; ++k) {
//					printf("\t\tL[%d][%d]: ", k, i);
//					L[k][i].print();
//					printf("\n");
//				}
//
//				for (int k = 0; k < 3; ++k) {
//					printf("\t\tU[%d][%d]: ", k, i);
//					U[k][i].print();
//					printf("\n");
//				}
//
//				printf("\t\tb[%d]: {%f, %f, %f}\n", i, b[i].x, b[i].y, b[i].z);
//				printf("\t\ty[%d]: {%f, %f, %f}\n", i, y[i].x, y[i].y, y[i].z);
//				printf("\t\tx[%d]: {%f, %f, %f}\n", i, vels[i].x, vels[i].y, vels[i].z);
//			}
//		}

//		if (si == 0) {
//			for (int i = si; i < numParticle; i += numStrand) {
//				float3 prevPos = prevPoses[i];
//				float3 restPos = restPoses[i];
//				float3 pos = poses[i];
//				float3 vel = vels[i];
//				float3 impulse =impulses[i];
//
//				printf("Particle(%d) {prevPos: {%f, %f, %f}, restPos: {%f, %f, %f}, pos: {%f, %f, %f}, vel: {%f, %f, %f}, impulse: {%f, %f, %f}, rigidness: %f}\n", i, prevPos.x, prevPos.y, prevPos.z, restPos.x, restPos.y, restPos.z, pos.x, pos.y, pos.z, vel.x, vel.y, vel.z, impulse.x, impulse.y, impulse.z, rigidness[i]);
//			}
//
//			printf("dTransform: "); dTransform.print(); printf("\n");
//			printf("dTranslation: {%f, %f, %f}\n", dTranslation.x, dTranslation.y, dTranslation.z);
//		}
	}

	void SelleMassSpringImplicitCudaSolver_resolveStrandDynamics(
			Mat3 **A,
			Mat3 **L,
			Mat3 **U,
			float3 *y,
			float3 *b,
			float3 *poses,
			float3 *prevPoses,
			float3 *restPoses,
			float3 *vels,
			float3 *impulses,
			const float *rigidness,
			Mat3 dTransform,
			float3 dTranslation,
			int numParticle,
			int numStrand,
			int numParticlePerStrand,
			float pmass,
			float damping,
			float kStretch,
			float kBending,
			float kTorsion,
			float strainLimitingTolerance,
			float t,
			int wrapSize
	) {
		// One for each strand
		int numThread = wrapSize * 8;
		int numBlock = (numStrand + numThread - 1) / numThread;

		SelleMassSpringImplicitCudaSolver_resolveStrandDynamicsKernal<<<numBlock, numThread>>>(
				A, L, U, y, b, poses, prevPoses, restPoses, vels, impulses,
				rigidness, dTransform, dTranslation, numParticle, numStrand, numParticlePerStrand,
				pmass, damping, kStretch, kBending, kTorsion, strainLimitingTolerance, t
		);

		hipDeviceSynchronize();
	}

	__global__
	void SelleMassSpringImplicitCudaSolver_getVelocityFromPositionKernal(
			const float3 * poses,
			const float3 * prevPoses,
			float3 * vels,
			float tInv,
			int numParticle
	) {
		// One for each particle
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= numParticle)
			return;

		vels[i] = (poses[i] - prevPoses[i]) * tInv;

//		if (i == 0) {
//			for (int k = 0; k < numParticle; ++k) {
//				float3 pos = poses[k];
//				float3 prevPos = prevPoses[k];
//				float3 vel = vels[k];
//				printf("%d: pos: {%f, %f, %f}, prevPos: {%f, %f, %f}, vel: {%f, %f, %f}\n", k, pos.x, pos.y, pos.z, prevPos.x, prevPos.y, prevPos.z, vel.x, vel.y, vel.z);
//			}
//		}
	}

	void SelleMassSpringImplicitCudaSolver_getVelocityFromPosition(
			const float3 * poses,
			const float3 * prevPoses,
			float3 * vels,
			float tInv,
			int numParticle,
			int wrapSize
	) {
		// One for each particle
		int numThread = wrapSize * 32;
		int numBlock = (numParticle + numThread - 1) / numThread;

		SelleMassSpringImplicitCudaSolver_getVelocityFromPositionKernal<<<numBlock, numThread>>>(
				poses, prevPoses, vels, tInv, numParticle
		);
		hipDeviceSynchronize();
	}
}