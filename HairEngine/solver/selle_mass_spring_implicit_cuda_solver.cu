#include "hip/hip_runtime.h"

#include "cudauntime.h"
#include "../util/simple_mat.h"
#include "selleasspringonf.h"
#include "selle_mass_spring_conf.h"

namespace HairEngine {

	__device__ __forceinline__
	void getSpringInfo(float3 p1, float3 p2, float3 rp1, float3 rp2, float t, float k, float3 &impulse, Mat3 &dm) {
		float3 d = p2 - p1;
		float l = length(d);
		d /= l;

		float f = t * t * k;

		impulse = (k * (l - length(rp1 - rp2)) * t) * d;

		dm.at<0, 0>() = f * d.x * d.x;
		dm.at<0, 1>() = dm.at<1, 0>() = f * d.x * d.y;
		dm.at<0, 2>() = dm.at<2, 0>() = f * d.x * d.z;
		dm.at<1, 1>() = f * d.y * d.y;
		dm.at<1, 2>() = dm.at<2, 1>() = f * d.y * d.z;
		dm.at<2, 2>() = f * d.z * d.z;
	}

	__global__
	void SelleMassSpringImplicitCudaSolver_resolveStrandDynamicsKernal(
			Mat3 **A,
			Mat3 **L,
			Mat3 **U,
			float3 *y,
			float3 *b,
			float3 *poses,
			float3 *prevPoses,
			float3 *restPoses,
			float3 *vels,
			float3 *impulses,
			const float *rigidness,
			Mat3 dTransform,
			float3 dTranslation,
			int numParticle,
			int numStrand,
			int numParticlePerStrand,
			float pmass,
			float damping,
			float kStretch,
			float kBending,
			float kTorsion,
			float strainLimitingTolerance,
			float t
	) {
		// One for each strand
		int si = blockIdx.x * blockDim.x + threadIdx.x;
		if (si >= numStrand)
			break;

		// Alias
		const auto & n = numParticlePerStrand;

		// Initialize the b and A
		float3 p[4]; // Previous poses, store in register
		float3 rp[4]; // Previous rest poses, store in register

		float3 impulse;
		Mat3 dm;
		for (int i = si, li = 0; i < numParticle; i += numStrand, ++li) {

			A[3][i] = Mat3::Diagnoal(pmass + damping * t);
			A[0][i] = A[1][i] = A[2][i] = A[4][i] = A[5][i] = A[6][i] = Mat3::Zero();

			b[i] = pmass * vels[i] + impulses[i] * t;

			// Apply spring force, since b[pi] and A[xxx][pi] has been correctly initialized, it is safe to modify
			p[3] = poses[i];
			rp[3] = restPoses[i];

			// Stretch spring
			if (li >= 1) {
				getSpringInfo(p[2], p[3], rp[2], rp[3], t, kStretch, impulse, dm);

				auto pi = i - numStrand;
				b[pi] += impulse;
				b[i] -= impulse;

				A[3][pi] += dm;
				A[3][i] += dm;
				A[4][pi] -= dm;
				A[2][i] -= dm;
			}

			// Bending spring
			if (li >= 2) {
				getSpringInfo(p[1], p[3], rp[1], rp[3], t, kBending, impulse, dm);

				auto pi = i - 2 * numStrand;
				b[pi] += impulse;
				b[i] -= impulse;

				A[3][pi] += dm;
				A[3][i] += dm;
				A[5][pi] -= dm;
				A[1][i] -= dm;
			}

			// Torsion spring
			if (li >= 3) {
				getSpringInfo(p[0], p[3], rp[0], rp[3], t, kTorsion, impulse, dm);

				auto pi = i - 3 * numStrand;
				b[pi] += impulse;
				b[i] -= impulse;

				A[3][pi] += dm;
				A[3][i] += dm;
				A[6][pi] -= dm;
				A[0][i] -= dm;
			}

			p[0] = p[1]; p[1] = p[2]; p[2] = p[3];
			rp[0] = rp[1]; rp[1] = rp[2]; rp[2] = rp[3];
		}

		// Initialize b and A of the strand root
		b[si] = dTransform * prevPoses[si] + dTranslation;
		A[0][si] = A[1][si] = A[2][si] = A[4][si] = A[5][si] = Mat3::Zero();
		A[3][si] = Mat3::Identity();

		// Heptadignoal solver
		for (int i = si, li = 0; i < numParticle; i += numStrand, ++li) {
			//compute L3
			L[3][i] = li >= 3 ? A[0][i] : Mat3::Zero();

			//compute L2
			L[2][i] = li >= 2 ? A[1][i] : Mat3::Zero();
			if (li >= 3)
				L[2][i] -= L[3][i] * U[0][i - 3];

			//compute L1
			L[1][i] = li >= 1 ? A[2][i] : Mat3::Zero();
			if (li >= 2)
				L[1][i] -= L[2][i] * U[0][i - 2];
			if (li >= 3)
				L[1][i] -= L[3][i] * U[1][i - 3];

			//compute L0
			L[0][i] = A[3][i];
			if (li >= 1)
				L[0][i] -= L[1][i] * U[0][i - 1];
			if (li >= 2)
				L[0][i] -= L[2][i] * U[1][i - 2];
			if (li >= 3)
				L[0][i] -= L[3][i] * U[2][i - 3];

			//compute L0i
			L[4][i] = L[0][i].inverse();

			//compute U2
			U[2][i] = li + 3 < n ? L[4][i] * A[6][i] : Mat3::Zero();

			//compute U1
			U[1][i] = li + 2 < n ? A[5][i] : Mat3::Zero();
			if (li >= 1)
				U[1][i] -= L[1][i] * U[2][i - 1];
			U[1][i] = L[4][i] * U[1][i];

			//compute U0
			U[0][i] = li + 1 < n ? A[4][i] : Mat3::Zero();
			if (li >= 1)
				U[0][i] -= L[1][i] * U[1][i - 1];
			if (li >= 2)
				U[0][i] -= L[2][i] * U[2][i - 2];
			U[0][i] = L[4][i] * U[0][i];

			//compute y
			y[i] = b[i];
			if (li >= 1)
				y[i] -= L[1][i] * y[i - 1];
			if (li >= 2)
				y[i] -= L[2][i] * y[i - 2];
			if (li >= 3)
				y[i] -= L[3][i] * y[i - 3];
			y[i] = L[4][i] * y[i];
		}

		// Compute the final velocity and poses
		for (int i = numParticle - numStrand + si, li = n - 1; i >= 0; i -= numStrand, --li) {
			vels[i] = y[i];
			if (li + 1 < n)
				vels[i] -= U[0][i] * x[i + 1];
			if (li + 2 < n)
				vels[i] -= U[1][i] * x[i + 2];
			if (li + 3 < n)
				vels[i] -= U[2][i] * x[i + 3];

			float3 prevPos = prevPoses[i];

			// Rigidness
			poses[i] = lerp(prevPos + vels[i] * t, dTransform * prevPos + dTranslation, rigidness[i]);
		}

		// Apply strain limiting
		p[0] = poses[si];
		rp[0] = restPoses[si];
		for (int i = si + numStrand; i < numParticle; i += numStrand) {
			p[1] = poses[i];
			rp[1] = restPoses[i];

			float ltol = length(rp[1] - rp[0]) * strainLimitingTolerance;

			float3 d = p[1] - p[0];
			float l = length(d);

			if (l > ltol)
				p[1] = p[0] + d * (ltol / l);

			// Write back
			poses[i] = p[1];

			// Assign for next itereation
			p[0] = p[1];
			rp[0] = rp[1];
		}
	}

	void SelleMassSpringImplicitCudaSolver_resolveStrandDynamics(
			Mat3 **A,
			Mat3 **L,
			Mat3 **U,
			float3 *y,
			float3 *b,
			float3 *poses,
			float3 *prevPoses,
			float3 *restPoses,
			float3 *vels,
			float3 *impulses,
			const float *rigidness,
			Mat3 dTransform,
			float3 dTranslation,
			int numParticle,
			int numStrand,
			int numParticlePerStrand,
			float pmass,
			float damping,
			float kStretch,
			float kBending,
			float kTorsion,
			float strainLimitingTolerance,
			float t,
			int wrapSize
	) {
		// One for each strand
		int numThread = wrapSize * 8;
		int numBlock = (numStrand + numThread - 1) / numThread;

		SelleMassSpringImplicitCudaSolver_resolveStrandDynamicsKernal<<<numBlock, numThread>>>(
				A, L, U, y, b, poses, prevPoses, restPoses, vels, impulses,
				rigidness, dTransform, dTranslation, numParticle, numStrand, numParticlePerStrand,
				pmass, damping, kStretch, kBending, kTorsion, strainLimitingTolerance, t
		);

		hipDeviceSynchronize();
	}

	__global__
	void SelleMassSpringImplicitCudaSolver_getVelocityFromPositionKernal(
			const float3 * poses,
			const float3 * prevPoses,
			float3 * vels,
			float tInv,
			int numParticle
	) {
		// One for each particle
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= numParticle)
			return;
		vels[i] = (poses[i] - prevPoses[i]) * tInv;
	}

	void SelleMassSpringImplicitCudaSolver_getVelocityFromPosition(
			const float3 * poses,
			const float3 * prevPoses,
			float3 * vels,
			float tInv,
			int numParticle,
			int wrapSize
	) {
		// One for each particle
		int numThread = wrapSize * 32;
		int numBlock = (numParticle + numThread - 1) / numThread;

		SelleMassSpringImplicitCudaSolver_getVelocityFromPositionKernal<<<numBlock, numThread>>>(
				poses, prevPoses, vels, tInv, numParticle
		);

		hipDeviceSynchronize();
	}
}